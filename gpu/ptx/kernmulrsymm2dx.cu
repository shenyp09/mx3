
#include <hip/hip_runtime.h>
// 2D X (out-of-plane only) micromagnetic kernel multiplication:
// Mx = Kxx * Mx
//
// ~kernel has mirror symmetry along Y-axis,
// apart form first row,
// and is only stored (roughly) half:
//
// K00:
// xxxxx
// aaaaa
// bbbbb
// ....
// bbbbb
// aaaaa
//
extern "C" __global__ void 
kernmulRSymm2Dx(float* fftMx, float* fftKxx, int N1, int N2){

	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.x * blockDim.x + threadIdx.x;

	if(j>= N1 || k>=N2){
 		return;	
	}

	int I = j*N2 + k;       // linear index for upper half of kernel
	int I2 = (N1-j)*N2 + k; // linear index for re-use of lower half

    float Kxx;

	if (j < N1/2 + 1){
		Kxx = fftKxx[I];
	}else{
		Kxx = fftKxx[I2];
	}

  	int e = 2 * I;

    float reMx = fftMx[e  ];
    float imMx = fftMx[e+1];

    fftMx[e  ] = reMx * Kxx;
    fftMx[e+1] = imMx * Kxx;
}
